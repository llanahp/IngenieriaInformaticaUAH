#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

const int N = 16;
const int M = 3;
const int width = 16;
const int TILE_WIDTH = 16 / 2;


__global__ void stencil(const int *a, const int *b, int*c)
{

	int fila = blockIdx.y*TILE_WIDTH + threadIdx.y;
	int columna = blockIdx.x*TILE_WIDTH + threadIdx.x;

	//Si no es de un borde
	if (columna != 0 && columna != N - 1 && fila != 0 && fila != N - 1) {
		
		int total = 0;
		int posicion = (fila*width + columna);
		//Calcula la columna de la izquierda
		int i = -1;
		total += a[posicion - width + i] * b[0] + a[posicion + i] * b[3] +a[posicion + width +i]* b[6];
		

		//Calcula la columna central
		
		total = total + (a[posicion - width ] * b[1]+ a[posicion] * b[4] + a[posicion + width ] * b[7] );

		
		//Calcula la columna de la derecha 
		i = 1;
		total = total + (a[posicion - width + i] * b[2] + a[posicion + i] * b[5] + a[posicion + width + i] * b[8]);
					
		
		//almaceno el resultado
		c[(fila - 1)*(width - 2) + columna - 1] = total;
	}
}

int main()
{
	//declaracion de matrices
	int  primeraMatrizH[N*N],segundaMatrizH[M*M], resultadoH[(N - 2)*(N - 2)];
	int* primeraMatrizD, *segundaMatrizD, *resultadoD;

	//size que van a tener las matrices
	int size = N * N* sizeof(int);

	//reservo memoria en Device
	hipMalloc((void **)&primeraMatrizD, size);
	hipMalloc((void **)&resultadoD, size);

	hipMalloc((void **)&segundaMatrizD, M*M * sizeof(int));
	
	//Declaracion de semilla aleatoria con la hora local para el generador de numeros aleatorios
	srand(time(NULL));

	//relleno las matrices
	
	for (int i = 0; i < N*N; i++) {	
		primeraMatrizH[i] = rand() % 256;
	}

	for (int i =0 ;i<M*M;i++) {
		segundaMatrizH[i] =rand() % 11;
	}
	int matrizTemporal[M*M];
	//cambio las posiciones del filtro
	for (int i = 0; i < M*M; i++) {
		
			matrizTemporal[i] = segundaMatrizH[abs(((M*M)-1)-i)];
	}
	

	//muestro las matrices por pantalla
	printf("Primera Matriz \n");
	for (int i = 0; i < N*N; i++) {
		if (i !=0 && i%16==0)
			printf("\n");
		printf("%d, ", primeraMatrizH[i]);
	}
	printf("\n");


	printf("\nSegunda Matriz \n");
	for (int i = 0; i < M*M; i++) {
		if (i != 0 && i % 3 == 0)
			printf("\n");
		printf("%d, ", segundaMatrizH[i]);
	}
	printf("\n");
	printf("\nSegunda Matriz transpuesta\n");
	for (int i = 0; i < M*M; i++) {
		if (i != 0 && i % 3 == 0)
			printf("\n");
		printf("%d, ", matrizTemporal[i]);
	}
	printf("\n\n");



	hipMemcpy(primeraMatrizD, &primeraMatrizH, size, hipMemcpyHostToDevice);
	hipMemcpy(segundaMatrizD, &matrizTemporal, M*M * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(resultadoD, &resultadoH, size, hipMemcpyHostToDevice);

	//cambio el valor en la GPU
	dim3 dimGrid(width / TILE_WIDTH, width / TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);


	//Stencil
	stencil << < dimGrid, dimBlock >> > (primeraMatrizD,segundaMatrizD, resultadoD);
	hipMemcpy(&resultadoH, resultadoD, size, hipMemcpyDeviceToHost);

	printf("Resultado\n");
	for (int i = 0; i < (N - 2)*(N - 2); i++) {
		if (i != 0 && i % 14 == 0)
			printf("\n");
		printf("%d, ", resultadoH[i]);
	}

	hipFree(primeraMatrizD);
	hipFree(resultadoD);

	//get char X2
	getchar();
	getchar();

	return 0;
}
