#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>


//Funcion auxiliar para el calculo de la integral
__device__ float Funcion(float x) {
    return (x / (x * x + 4)) * sin(1.0f / x);
}


__global__ void Integral(float* valor, int numTrapecios) {
    

    // La integral es de 1 a 3.
    // Cada trapecio tendrá una base de 2 / número de trapecios,
    // de tal manera que la suma de todas las bases sea 2.
    float base = 2.0f / numTrapecios;
    float x = 1 + threadIdx.x * base;

    // El tamaño del array se asignara dinamicamente durante ejecucion, que será el máximo numero de threads de un bloque
    extern __shared__ float areas[];

    // Área del trapecio: (altura del primer punto + altura del segundo punto) / 2.
    areas[threadIdx.x] = base * (Funcion(x) + Funcion(x + base)) / 2;

    __syncthreads();

    //Un único recorre el array sumando todos los valores de las areas para obtener el resultado
    if (x == 1)
        for (int i = 0; i < numTrapecios; i++) {
            *valor += areas[i];
        }
    __syncthreads();
}

int main() {

    //Obtenemos las propiedades de la tarjeta
    hipDeviceProp_t gpuProperties{};
    hipGetDeviceProperties(&gpuProperties, 0);

    //Obtenemos el maximo numero de threads por bloque:
    int maxNumThreads = gpuProperties.maxThreadsPerBlock;

    //Declaramos la dimension del array que despues se pasará en el kernel
    int dimArray = sizeof(int) * maxNumThreads;

    //Asignamos las variables para el resultado en el dispositivo y en la CPU
    float* resultado_dev;
    float resultado = 0;

    //Reservamos la memoria en el dispositivo
    hipMalloc(&resultado_dev, sizeof(float));

    //hipMemcpy(resultado_dev, &resultado, sizeof(float), hipMemcpyKind::hipMemcpyHostToDevice);

  

    //Creamos un grid con un bloque con el maximo numero de threads que nos permite la tarjeta
    dim3 dimGrid = dim3(1, 1);
    dim3 dimBlock = dim3(maxNumThreads, 1);
    
    //Invocamos la kernel con las dimensiones, la variable resultado y el maximo numero de Threads como parametros
    Integral << <dimGrid,dimBlock,dimArray >> > (resultado_dev, maxNumThreads);

    //Copiamos el resultado de vuelta de la tarjeta
    hipMemcpy(&resultado, resultado_dev, sizeof(float), hipMemcpyKind::hipMemcpyDeviceToHost);

    //Imprimimos el resultado
    printf("resultado: %f", resultado);

    //Liberamos las variables
    hipFree(resultado_dev);

    return 0;
}
