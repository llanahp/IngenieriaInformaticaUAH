#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <cassert>

#define MATRIX_WIDTH 16
constexpr int CONVOLUCION_WIDTH = 3;
constexpr int TILE_WIDTH = 16 / 2;

//Define para obtener la posicion global de la matriz
#define GET_INDEX(x, y, width) (y) * (width) + (x)

//Funcion para el calculo de la convolucion recibe la matriz para convolucion en matriz y en convolucion
__global__ void Convolucion(const int* matriz, const int* convolucion, int* resultado) {
	
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	//Calculamos el inicio y el final del bloque
	int startX = blockIdx.x * blockDim.x;
	int startY = blockIdx.y * blockDim.y;
	
	//El final es el primer elemento del siguiente bloque menos 1
	int endX = blockIdx.x * blockDim.x + blockDim.x - 1;
	int endY = blockIdx.y * blockDim.y + blockDim.y - 1;
	
	//Matriz compartida
	__shared__ int sharedMatriz[TILE_WIDTH][TILE_WIDTH];
	
	//Cada thread rellena su numero de la compartida
	sharedMatriz[threadIdx.x][threadIdx.y] = matriz[GET_INDEX(x, y, MATRIX_WIDTH)];

	__syncthreads();

	//Si no es de un borde
	if (x > 0 && x < MATRIX_WIDTH - 1 && y > 0 && y < MATRIX_WIDTH - 1) {
		int value = 0;
		//Cogemos los elementos de la matriz para la convolucion
		for (int diffX = -1; diffX < 1; diffX++) {
			
			for (int diffY = -1; diffY < 1; diffY++) {
				//Si esta dentro de los limites de la memoria compartida
				if (x + diffX >= startX && x + diffX <= endX && y + diffY >= startY && y + diffY <= endY)
					//Se 
					value += sharedMatriz[threadIdx.x + diffX][threadIdx.y + diffY] * convolucion[GET_INDEX(diffX + 1, diffY + 1, CONVOLUCION_WIDTH)];
				
				else
					value += matriz[GET_INDEX(x + diffX, y + diffY, MATRIX_WIDTH)] * convolucion[GET_INDEX(diffX + 1, diffY + 1, CONVOLUCION_WIDTH)];
				
			}
		}
		
		//Se resta 1 para comenzar desde la primera fila ya que la X empieza en la segunda columna
		resultado[GET_INDEX(x - 1, y - 1, MATRIX_WIDTH - 2)] = value;
	}

	__syncthreads();
}

int main()
{
	//declaracion de matrices
	int matriz[MATRIX_WIDTH * MATRIX_WIDTH];
	int convolucion[CONVOLUCION_WIDTH * CONVOLUCION_WIDTH];
	int resultado[(MATRIX_WIDTH - 2) * (MATRIX_WIDTH - 2)];

	int* matriz_dev = nullptr;
	int* convolucion_dev = nullptr;
	int* resultado_dev = nullptr;

	//reservo memoria en Device
	hipMalloc(&matriz_dev, sizeof(matriz));
	hipMalloc(&convolucion_dev, sizeof(convolucion));
	hipMalloc(&resultado_dev, sizeof(resultado));

	//Declaracion de semilla aleatoria con la hora local para el generador de numeros aleatorios
	srand(time(NULL));

	//relleno las matrices

	for (int i = 0; i < _countof(matriz); i++)
		matriz[i] = rand() % 256;

	for (int i = 0; i < _countof(convolucion); i++)
		convolucion[i] = rand() % 11;

	int matrizTemporal[CONVOLUCION_WIDTH * CONVOLUCION_WIDTH];
	//cambio las posiciones del filtro
	for (int i = 0; i < _countof(matrizTemporal); i++)
		matrizTemporal[i] = convolucion[abs(((CONVOLUCION_WIDTH * CONVOLUCION_WIDTH) - 1) - i)];


	//muestro las matrices por pantalla
	printf("Primera Matriz \n");
	for (int i = 0; i < _countof(matriz); i++) {
		if (i != 0 && i % 16 == 0)
			printf("\n");
		printf("%i, ", matriz[i]);
	}
	printf("\n");


	printf("\nSegunda Matriz \n");
	for (int i = 0; i < _countof(convolucion); i++) {
		if (i != 0 && i % 3 == 0)
			printf("\n");
		printf("%i, ", convolucion[i]);
	}
	printf("\n");
	printf("\nSegunda Matriz transpuesta\n");
	for (int i = 0; i < _countof(matrizTemporal); i++) {
		if (i != 0 && i % 3 == 0)
			printf("\n");
		printf("%i, ", matrizTemporal[i]);
	}
	printf("\n\n");



	hipMemcpy(matriz_dev, &matriz, sizeof(matriz), hipMemcpyHostToDevice);
	hipMemcpy(convolucion_dev, &matrizTemporal, sizeof(matrizTemporal), hipMemcpyHostToDevice);

	//cambio el valor en la GPU
	dim3 dimGrid(MATRIX_WIDTH / TILE_WIDTH, MATRIX_WIDTH / TILE_WIDTH);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);


	//Realizamos la convolucion
	Convolucion <<<dimGrid, dimBlock>>>(matriz_dev, convolucion_dev, resultado_dev);

	printf("Resultado size: %i\n", sizeof(resultado));
	hipMemcpy(&resultado, resultado_dev, sizeof(resultado), hipMemcpyDeviceToHost);

	printf("Resultado\n");
	for (int i = 0; i < _countof(resultado); i++) {
		if (i != 0 && i % 14 == 0)
			printf("\n");
		printf("%i, ", resultado[i]);
	}

	hipFree(matriz_dev);
	hipFree(resultado_dev);

	//get char X2
	getchar();
	getchar();

	return 0;
}
